#include "hip/hip_runtime.h"
#include "agent.cuh"
#include "octree.hxx"


int Agent::curr_state =0;


__device__ Agent::Agent(const Vector &pos, const Vector &vel, const Vector &dir, const int curr){
//TODO Use of position/velocity/direction lists ? + Parallelisr les opérations
//sur direction/velocity/position
  position[curr] = pos;
  velocity[curr] = vel;
  direction[curr] = dir;
}

Agent::Agent(const Vector &pos, const Vector &vel, const Vector &dir){
//TODO Use of position/velocity/direction lists ? + Parallelisr les opérations
//sur direction/velocity/position
  position[Agent::curr_state] = pos;
  velocity[Agent::curr_state] = vel;
  direction[Agent::curr_state] = dir;
}

__host__ __device__ Agent::Agent(){
}



__device__  Vector  Agent::separation(Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

   Vector force;
    int count =0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (this->position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force -= (this->position[curr] - agent_list[i].position[curr]).normalized();
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
//return force;
}


__device__ Vector Agent::cohesion(Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (this->position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].position[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}

__device__ Vector Agent::alignment(Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {
    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].velocity[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}


/*size_t Agent::find_closest(Container &agent_list, size_t index) {
  size_t closest_agent = index;
  double min_dist = 1000;

  double dist;

  for(size_t i = 0; i < agent_list.size(); i++) {
    if (i != index) {
      dist= (this->position[this->curr_state] - agent_list[i].position[this->curr_state]).norm();

      if(dist < min_dist) {
        min_dist = dist;
        closest_agent = i;
      }
    }
  }
  return closest_agent;
}*/

/* Return all the neighbours of an agent without itself */









