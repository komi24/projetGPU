#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <ctime>
#include <omp.h>

#include "workspace.hxx"
#include "agent.cuh"
#include "vector.cuh"
#include "tester.hxx"
//#include "octree.hxx"

#include <hip/hip_runtime.h>

#define BUFF_SIZE 20

Workspace::Workspace(ArgumentParser &parser)
{

  na = parser("agents").asInt();

  wCohesion = parser("wc").asDouble();
  wAlignment = parser("wa").asDouble();
  wSeparation = parser("ws").asDouble();

  rCohesion = parser("rc").asDouble();
  rAlignment = parser("ra").asDouble();
  rSeparation = parser("rs").asDouble();
  dt= 0.05;
  maxU = 2.0;
  time = 0.,//;

  this->init();}

Workspace::Workspace(size_t nAgents,
             Real wc, Real wa, Real ws,
             Real rc, Real ra, Real rs) :
             na(nAgents), dt(.05), time(0),
             wCohesion(wc), wAlignment(wa), wSeparation(ws),
             rCohesion(rc), rAlignment(ra), rSeparation(rs),
             maxU(2.)
{ this->init();}

void  Workspace::init(){
    domainsize = 1.0;

    // Random generator seed
    srand48(std::time(0));

    //Initializing Octree head
    Real maxR;
    maxR = (rCohesion > rSeparation) ? rCohesion : rSeparation;
    maxR = (maxR > rAlignment) ? maxR : rAlignment;
    oc = *(new Octree(2*maxR,domainsize));

    // Initialize agents
    // This loop may be quite expensive due to random number generation
    //agents.reserve(na);
    //#pragma omp parallel
    //{
      //#pragma omp for
      for(size_t j = 0; j < na; j++){
      // Create random position
        Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      
	Agent *agt = new Agent(position,Zeros(),Zeros());
        oc.add(*agt);
      }
    //}

    /*for(size_t j = 0; j < na; j++){
      // Create random position
      Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      //agents.push_back(Agent(position, Zeros(), Zeros()));
      //agents.assign(j,Agent(position, Zeros(), Zeros()));
      oc.add(agents[j]);
    }*/

    /* TODO build the octree */

}

Agent *Workspace::tempToArray(TemporaryContainer tp){
 std::cerr << " CPU" << std::endl;
  Agent *res = (Agent*) malloc(tp.size()*sizeof(Agent));
  for(int i =0; i<tp.size(); i++){
    res[i]=*tp[i];
    //std::cerr << tp[i]->position[Agent::curr_state].x << " CPU" << std::endl;
   
  }

  return res;
}

void Workspace::arrayToTemp(Agent *agts, int s,TemporaryContainer &leaf){
  leaf.clear();
 std::cerr << " GPU" << std::endl;
  for(int i =0; i<s; i++)
  {
    leaf.push_back(&agts[i]);
    //std::cerr << agts[i].position[Agent::curr_state].x << " GPU" << std::endl;
    //std::cerr << agts[i].position[1- Agent::curr_state].x << " 1- curr GPU" << std::endl;
   
  }
    
}



__device__  Vector separation(Agent &a, Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

    Vector force = Vector();
    int count =0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (a.position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force -= (a.position[curr] - agent_list[i].position[curr]).normalized();
            ++count;
        }
    }
    return ( count >0 ? force/count : force);

}


__device__ Vector cohesion(Agent &a,Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (a.position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].position[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}

__device__ Vector alignment(Agent &a,Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {
    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].velocity[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}

__global__ void computeOnGPU(int sizeNb, int sizeLf, 
        Agent *agts, Agent *neigh, 
        Real rs, Real rc, Real ra, 
        Real wSeparation, Real wCohesion, Real wAlignment, 
        int curr, Real maxU,Real dt){
    int tileWidth = sizeNb/sizeLf;
    __shared__ Real ds_neighInst[BUFF_SIZE*sizeof(Agent)/sizeof(Real)];//TODO mettre à zero les champs
    __shared__ Agent *ds_neigh;
    ds_neigh = (Agent *) ds_neighInst;
    //ds_neigh = (Agent*) malloc(sizeof(Agent)*(tileWidth)); // TODO Faire gaffe un seul thread
    __shared__ Real ds_dist[BUFF_SIZE];
    //ds_dist = (Real *) malloc(sizeof(Real)*(tileWidth));
    Vector s, c, a;

    for (int j=0; j<sizeLf; j++){
        //Chargement mémoire
        for (int i= 0; i<tileWidth; i++){
            ds_neigh[i]=neigh[(blockIdx.x+j)*tileWidth+i];
        }
        __syncthreads();
        //Calcul des distances
        for (int i= 0; i<tileWidth; i++){
            ds_dist[i]=(agts[blockIdx.x].position[curr]-ds_neigh[(blockIdx.x+j)*tileWidth+i].position[curr]).norm();//TODO passer norm en __global__
        }
        __syncthreads();
        //Calcul des forces 
        //s =
        s =  separation(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, rs, curr);
        c = cohesion(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, rc, curr);
        a = alignment(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, ra, curr);

        agts[blockIdx.x].direction[1-curr] = c*wCohesion + a*wAlignment + s*wSeparation;

        agts[blockIdx.x].velocity[1-curr] = agts[blockIdx.x].velocity[curr] 
            + agts[blockIdx.x].direction[1-curr];
        float speed =agts[blockIdx.x].velocity[1-curr].norm();
        if ((speed > maxU)) {
            agts[blockIdx.x].velocity[1-curr] = agts[blockIdx.x].velocity[1-curr] * maxU/speed;
        }
        agts[blockIdx.x].position[1-curr] = agts[blockIdx.x].position[curr] + agts[blockIdx.x].velocity[curr]*dt;

        __syncthreads();
    }
  
}

void Workspace::move(int step)//TODO erase step (just for tests)
{

  Vector s,c,a;  
  LeafContainer leafs = Octree::leafs;
  TemporaryContainer nb;
  //std::cout << " leaves "<< Octree::leafs.size() << std::endl;
  for (size_t i=0; i<leafs.size(); i++){
    Octree *it=leafs[i];
    nb.clear();
    (it)->returnNeighboursLeaf(nb);
    TemporaryContainer agentsleaf = (it)->agents;

    //Chargement mémoire sur GPU
    Agent *neighArray=tempToArray(nb);
    Agent *leafArray=tempToArray(leafs[i]->agents);
 
    Agent *d_neighArray;
    Agent *d_leafArray;

    //TODO penser à supprimer les liste d'agents copiées
    hipMalloc((void **)&d_neighArray,sizeof(Agent)*nb.size());
    hipMalloc((void **)&d_leafArray,sizeof(Agent)*leafs[i]->agents.size());
    hipMemcpy(d_neighArray,neighArray,sizeof(Agent)*nb.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_leafArray,leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyHostToDevice);
    
    //Initialiser la grille
    dim3 dimGrid(leafs[i]->agents.size(),1,1);
    dim3 dimBlock(1,1,1);

    computeOnGPU<<<dimGrid,dimBlock>>>(nb.size(), leafs[i]->agents.size(), 
        d_leafArray, d_neighArray, 
         rSeparation,  rCohesion,  rAlignment, 
         wSeparation,  wCohesion,  wAlignment, 
         Agent::curr_state,maxU,dt);
    hipError_t err = hipGetLastError();
    if(hipSuccess != err )
        std::cerr << hipGetErrorString(err) << std::endl;
    hipDeviceSynchronize();
    hipMemcpy(leafArray,d_leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyDeviceToHost);

  
    arrayToTemp(leafArray,leafs[i]->agents.size(),leafs[i]->agents);

    hipFree(d_neighArray);
    //hipFree(neighArray);
    hipFree(d_leafArray);
    //hipFree(leafArray);

    hipDeviceSynchronize();

      for(size_t j=0; j<agentsleaf.size(); j++){

       Agent *it2=leafs[i]->agents[j];

      (it2)->position[1-Agent::curr_state].x= fmod((it2)->position[1-Agent::curr_state].x,domainsize);
      (it2)->position[1-Agent::curr_state].y= fmod((it2)->position[1-Agent::curr_state].y,domainsize);
      (it2)->position[1-Agent::curr_state].z= fmod((it2)->position[1-Agent::curr_state].z,domainsize);

    }

    }

    Agent::curr_state = 1 - Agent::curr_state;
    std::cerr << "ok1" << std::endl;
    update();
    std::cerr << "ok2" << std::endl;
}

void Workspace::returnNeighboursBuffer(TemporaryContainer &nb, Agent *agent,
  Real rc, TemporaryContainer &bufC,
  Real ra, TemporaryContainer &bufA,
  Real rs, TemporaryContainer &bufS
  ){
  for(int i=0; i<nb.size(); i++){
    Real dist =  (agent->position[Agent::curr_state] - nb[i]->position[Agent::curr_state]).norm();
    if(dist <= rc)
      bufC.push_back(agent);
    if(dist <= ra)
      bufA.push_back(agent);
    if(dist <= rs)
      bufS.push_back(agent);
  }  
}



void Workspace::update(){
  //#pragma omp parallel for
  LeafContainer leafs = Octree::leafs;

   for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++){
        if ((lf->position > lf->agents[j]->position[Agent::curr_state]) ||
         (lf->agents[j]->position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))){      
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          lf->agents[j]));
        //lf->agents.remove(&agents[k]);
        oc.add(*lf->agents[j]);
            std::cerr << "test3" << std::endl;
        lf->delete_leaves();
            std::cerr << "test4" << std::endl;
      } else {
            std::cerr << "test5" << std::endl;
         lf->agents[j]->leaf[Agent::curr_state]=lf;
            std::cerr << "test5" << std::endl;
      }   
    }
      }

  /*for(size_t k = 0; k< na; k++){
    if (Agent::curr_state)
    Octree *lf = agents[k].leaf[1-Agent::curr_state];
    //Retirer de la liste si nécessaire et rajouter au bon endroit
    if((lf->position > agents[k].position[Agent::curr_state]) 
      || (agents[k].position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))) {
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          &agents[k]));
        //lf->agents.remove(&agents[k]);
        lf->delete_leaves();
        oc.add(agents[k]);
      } else {
        agents[k].leaf[Agent::curr_state]=lf;
      }
    }*/
  }


void Workspace::simulate(int nsteps) {
  // store initial position[Agent::curr_state]s
    save(0);

    // perform nsteps time steps of the simulation
    int step = 0;
    while (step++ < nsteps) {
    //std::cout << "coco" << step << std::endl; 
      this->move(step);
    std::cerr << "ok3" << std::endl;
      //tst.printOctree(& this->oc);
      // store every 20 steps
      if (step%1 == 0) save(step);
    }
}

void Workspace::save(int stepid) {
  std::ofstream myfile;
  LeafContainer leafs=Octree::leafs;
    std::cerr << "ok4" << std::endl;
  myfile.open("boids.xyz", stepid==0 ? std::ios::out : std::ios::app);
    std::cerr << "ok4" << std::endl;

    myfile << std::endl;
    myfile << na << std::endl;
    for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++)
        myfile << "B " << lf->agents[j]->position[Agent::curr_state];
}
    myfile.close();
}

