#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <ctime>
#include <omp.h>

#include "workspace.hxx"
#include "agent.cuh"
#include "vector.cuh"
#include "tester.hxx"
//#include "octree.hxx"

#include <hip/hip_runtime.h>


Workspace::Workspace(ArgumentParser &parser)
{

  na = parser("agents").asInt();

  wCohesion = parser("wc").asDouble();
  wAlignment = parser("wa").asDouble();
  wSeparation = parser("ws").asDouble();

  rCohesion = parser("rc").asDouble();
  rAlignment = parser("ra").asDouble();
  rSeparation = parser("rs").asDouble();
  dt= 0.05;
  maxU = 2.0;
  time = 0.,//;

  this->init();}

Workspace::Workspace(size_t nAgents,
             Real wc, Real wa, Real ws,
             Real rc, Real ra, Real rs) :
             na(nAgents), dt(.05), time(0),
             wCohesion(wc), wAlignment(wa), wSeparation(ws),
             rCohesion(rc), rAlignment(ra), rSeparation(rs),
             maxU(2.)
{ this->init();}

void  Workspace::init(){
    domainsize = 1.0;

    // Random generator seed
    srand48(std::time(0));

    //Initializing Octree head
    Real maxR;
    maxR = (rCohesion > rSeparation) ? rCohesion : rSeparation;
    maxR = (maxR > rAlignment) ? maxR : rAlignment;
    oc = *(new Octree(2*maxR,domainsize));

    // Initialize agents
    // This loop may be quite expensive due to random number generation
    //agents.reserve(na);
    //#pragma omp parallel
    //{
      //#pragma omp for
      for(size_t j = 0; j < na; j++){
      // Create random position
        Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      
	Agent *agt = new Agent(position,Zeros(),Zeros());
        oc.add(*agt);
      }
    //}

    /*for(size_t j = 0; j < na; j++){
      // Create random position
      Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      //agents.push_back(Agent(position, Zeros(), Zeros()));
      //agents.assign(j,Agent(position, Zeros(), Zeros()));
      oc.add(agents[j]);
    }*/

    /* TODO build the octree */

}

Agent *Workspace::tempToArray(TemporaryContainer tp){
 std::cerr << " CPU" << std::endl;
  Agent *res = (Agent*) malloc(tp.size()*sizeof(Agent));
  for(int i =0; i<tp.size(); i++){
    res[i]=*tp[i];
    std::cerr << tp[i]->position[0].x << " CPU" << std::endl;
  }

  return res;
}

void Workspace::arrayToTemp(Agent *agts, int s,TemporaryContainer &leaf){
  leaf.clear();
 std::cerr << " GPU" << std::endl;
  for(int i =0; i<s; i++)
  {
    leaf.push_back(&agts[i]);
    std::cerr << agts[i].position[1-Agent::curr_state].x << " GPU" << std::endl;
  }
    
}



__device__  Vector separation(Agent &a, Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

    Vector force = Vector();
    int count =0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (a.position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force -= (a.position[curr] - agent_list[i].position[curr]).normalized();
            ++count;
        }
    }
    return ( count >0 ? force/count : force);

}


__device__ Vector cohesion(Agent &a,Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {

    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        //double dist = (a.position[this->curr_state] - agent_list[i]->position[this->curr_state]).norm();
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].position[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}

__device__ Vector alignment(Agent &a,Agent *agent_list, int sizeNeigh, Real *dist, Real rad, int curr) {
    Vector force;
    int count = 0;
    for(size_t i = 0; i < sizeNeigh; i++) {
        if ((dist[i] < rad) && (0<dist[i])) {
            // TODO the comparison is no longer needed //
            force += agent_list[i].velocity[curr];
            ++count;
        }
    }
    return ( count >0 ? force/count : force);
}

__global__ void computeOnGPU(int sizeNb, int sizeLf, 
        Agent *agts, Agent *neigh, 
        Real rs, Real rc, Real ra, 
        Real wSeparation, Real wCohesion, Real wAlignment, 
        int curr, Real maxU,Real dt){

    int tileWidth = sizeNb/sizeLf;
    __shared__ Agent *ds_neigh;
    ds_neigh = (Agent*) malloc(sizeof(Agent)*(tileWidth)); // Faire gaffe un seul thread
    __shared__ Real *ds_dist;
    ds_dist = (Real *) malloc(sizeof(Real)*(tileWidth));
    Vector s, c, a;

    //Chargement mémoire
    for (int i= 0; i<tileWidth; i++){
        ds_neigh[i]=neigh[blockIdx.x*tileWidth+i];
    }
    __syncthreads();
    //Calcul des distances
    for (int i= 0; i<tileWidth; i++){
        ds_dist[i]=(agts[blockIdx.x].position[curr]-ds_neigh[blockIdx.x*tileWidth+i].position[curr]).norm();//TODO passer norm en __global__
    }
    __syncthreads();
    //Calcul des forces 
    //s =
    s =  separation(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, rs, curr);
    c = cohesion(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, rc, curr);
    a = alignment(agts[blockIdx.x],ds_neigh,tileWidth, ds_dist, ra, curr);
    agts[blockIdx.x].direction[1-curr] = c*wCohesion + a*wAlignment + s*wSeparation;

    agts[blockIdx.x].velocity[1-curr] = agts[blockIdx.x].velocity[curr] 
        + agts[blockIdx.x].direction[1-curr];
     float speed =agts[blockIdx.x].velocity[1-curr].norm();
       if ((speed > maxU)) {
         agts[blockIdx.x].velocity[1-curr] = agts[blockIdx.x].velocity[1-curr] * maxU/speed;
      }
      agts[blockIdx.x].position[1-curr] = agts[blockIdx.x].position[curr] + agts[blockIdx.x].velocity[curr]*dt;
    __syncthreads();
  
}

void Workspace::move(int step)//TODO erase step (just for tests)
{
  Vector s,c,a;  
  LeafContainer leafs = Octree::leafs;
  TemporaryContainer nb;
  //std::cout << " leaves "<< Octree::leafs.size() << std::endl;
  for (size_t i=0; i<leafs.size(); i++){
    Octree *it=leafs[i];
    nb.clear();
    (it)->returnNeighboursLeaf(nb);
    TemporaryContainer agentsleaf = (it)->agents;

    //Chargement mémoire sur GPU
    Agent *neighArray=tempToArray(nb);
    Agent *leafArray=tempToArray(leafs[i]->agents);

    Agent *d_neighArray;
    Agent *d_leafArray;

    //TODO penser à supprimer les liste d'agents copiés
    hipMalloc((void **)&d_neighArray,sizeof(Agent)*nb.size());
    hipMalloc((void **)&d_leafArray,sizeof(Agent)*leafs[i]->agents.size());
    hipMemcpy(d_neighArray,neighArray,sizeof(Agent)*nb.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_leafArray,leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyHostToDevice);
    
    //Initialiser la grille
    dim3 dimGrid(leafs[i]->agents.size(),1,1);
    dim3 dimBlock(1,1,1);

    computeOnGPU<<<dimGrid,dimBlock>>>(nb.size(), leafs[i]->agents.size(), 
        d_leafArray, d_neighArray, 
         rSeparation,  rCohesion,  rAlignment, 
         wSeparation,  wCohesion,  wAlignment, 
         Agent::curr_state,maxU,dt);
    hipDeviceSynchronize();
    hipMemcpy(leafArray,d_leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyDeviceToHost);
  
    arrayToTemp(leafArray,leafs[i]->agents.size(),leafs[i]->agents);

      for(size_t j=0; j<agentsleaf.size(); j++){
       Agent *it2=agentsleaf[j];
      /* TemporaryContainer bufA,bufC,bufS;
       returnNeighboursBuffer(nb, it2,
        rCohesion, bufC,
        rAlignment, bufA,
        rSeparation, bufS);
       s = (it2)->separation(bufS, rSeparation);
       c = (it2)->cohesion(bufC, rCohesion);
       a = (it2)->alignment(bufA, rAlignment);
       (it2)->direction[1-Agent::curr_state] = wCohesion*c + wAlignment*a + wSeparation*s;

       (it2)->velocity[1-Agent::curr_state] = (it2)->velocity[Agent::curr_state] + (it2)->direction[1-Agent::curr_state];

       double speed = (it2)->velocity[1-Agent::curr_state].norm();
       if ((speed > maxU)) {
          (it2)->velocity[1-Agent::curr_state] = (it2)->velocity[1-Agent::curr_state] * maxU/speed;
      }

      (it2)->position[1-Agent::curr_state] = (it2)->position[Agent::curr_state] + dt*(it2)->velocity[Agent::curr_state];
*/
      (it2)->position[1-Agent::curr_state].x= fmod((it2)->position[1-Agent::curr_state].x,domainsize);
      (it2)->position[1-Agent::curr_state].y= fmod((it2)->position[1-Agent::curr_state].y,domainsize);
      (it2)->position[1-Agent::curr_state].z= fmod((it2)->position[1-Agent::curr_state].z,domainsize);

    }

    }

    Agent::curr_state = 1 - Agent::curr_state;
    update();
}

void Workspace::returnNeighboursBuffer(TemporaryContainer &nb, Agent *agent,
  Real rc, TemporaryContainer &bufC,
  Real ra, TemporaryContainer &bufA,
  Real rs, TemporaryContainer &bufS
  ){
  for(int i=0; i<nb.size(); i++){
    Real dist =  (agent->position[Agent::curr_state] - nb[i]->position[Agent::curr_state]).norm();
    if(dist <= rc)
      bufC.push_back(agent);
    if(dist <= ra)
      bufA.push_back(agent);
    if(dist <= rs)
      bufS.push_back(agent);
  }  
}



void Workspace::update(){
  //#pragma omp parallel for
  LeafContainer leafs = Octree::leafs;

   for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++){
        if ((lf->position > lf->agents[j]->position[Agent::curr_state]) ||
         (lf->agents[j]->position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))){      
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          lf->agents[j]));
        //lf->agents.remove(&agents[k]);
        lf->delete_leaves();
        oc.add(*lf->agents[j]);
      } else {
         lf->agents[j]->leaf[Agent::curr_state]=lf;
      }   
    }
      }

  /*for(size_t k = 0; k< na; k++){
    if (Agent::curr_state)
    Octree *lf = agents[k].leaf[1-Agent::curr_state];
    //Retirer de la liste si nécessaire et rajouter au bon endroit
    if((lf->position > agents[k].position[Agent::curr_state]) 
      || (agents[k].position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))) {
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          &agents[k]));
        //lf->agents.remove(&agents[k]);
        lf->delete_leaves();
        oc.add(agents[k]);
      } else {
        agents[k].leaf[Agent::curr_state]=lf;
      }
    }*/
  }


void Workspace::simulate(int nsteps) {
  // store initial position[Agent::curr_state]s
    save(0);

    // perform nsteps time steps of the simulation
    int step = 0;
    while (step++ < nsteps) {
    //std::cout << "coco" << step << std::endl; 
      this->move(step);
      //tst.printOctree(& this->oc);
      // store every 20 steps
      if (step%1 == 0) save(step);
    }
}

void Workspace::save(int stepid) {
  std::ofstream myfile;
  LeafContainer leafs=Octree::leafs;
  myfile.open("boids2.xyz", stepid==0 ? std::ios::out : std::ios::app);

    myfile << std::endl;
    myfile << na << std::endl;
    for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++)
        myfile << "B " << lf->agents[j]->position[Agent::curr_state];
}
    myfile.close();
}

