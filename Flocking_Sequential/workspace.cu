#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <ctime>
#include <omp.h>

#include "workspace.hxx"
#include "agent.hxx"
#include "vector.hxx"
#include "tester.hxx"
//#include "octree.hxx"

#include <hip/hip_runtime.h>


Workspace::Workspace(ArgumentParser &parser)
{

  na = parser("agents").asInt();

  wCohesion = parser("wc").asDouble();
  wAlignment = parser("wa").asDouble();
  wSeparation = parser("ws").asDouble();

  rCohesion = parser("rc").asDouble();
  rAlignment = parser("ra").asDouble();
  rSeparation = parser("rs").asDouble();
  dt= 0.05;
  maxU = 2.0;
  time = 0.,//;

  this->init();}

Workspace::Workspace(size_t nAgents,
             Real wc, Real wa, Real ws,
             Real rc, Real ra, Real rs) :
             na(nAgents), dt(.05), time(0),
             wCohesion(wc), wAlignment(wa), wSeparation(ws),
             rCohesion(rc), rAlignment(ra), rSeparation(rs),
             maxU(2.)
{ this->init();}

void  Workspace::init(){
    domainsize = 1.0;

    // Random generator seed
    srand48(std::time(0));

    //Initializing Octree head
    Real maxR;
    maxR = (rCohesion > rSeparation) ? rCohesion : rSeparation;
    maxR = (maxR > rAlignment) ? maxR : rAlignment;
    oc = *(new Octree(2*maxR,domainsize));

    // Initialize agents
    // This loop may be quite expensive due to random number generation
    //agents.reserve(na);
    //#pragma omp parallel
    //{
      //#pragma omp for
      for(size_t j = 0; j < na; j++){
      // Create random position
        Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      
	Agent *agt = new Agent(position,Zeros(),Zeros());
        oc.add(*agt);
      }
    //}

    /*for(size_t j = 0; j < na; j++){
      // Create random position
      Vector position(drand48(), drand48(), drand48());

      // Create random velocity
      //agents.push_back(Agent(position, Zeros(), Zeros()));
      //agents.assign(j,Agent(position, Zeros(), Zeros()));
      oc.add(agents[j]);
    }*/

    /* TODO build the octree */

}

Agent *Workspace::tempToArray(TemporaryContainer tp){
 std::cerr << " CPU" << std::endl;
  Agent *res = (Agent*) malloc(tp.size()*sizeof(Agent));
  for(int i =0; i<tp.size(); i++){
    res[i]=*tp[i];
    std::cerr << tp[i]->position[0].x << " CPU" << std::endl;
  }

  return res;
}

void Workspace::arrayToTemp(Agent *agts, int s,TemporaryContainer &leaf){
  leaf.clear();
 std::cerr << " GPU" << std::endl;
  for(int i =0; i<s; i++)
  {
    leaf.push_back(&agts[i]);
    std::cerr << agts[i].position[0].x << " GPU" << std::endl;
  }
    
}


__global__ void computeOnGPU(Real r, Agent *agts){
  int *t = (int*) agts;
  for (int i=0; i<sizeof(Agent)/sizeof(int); i++)
      t[i]=0;
  
}

void Workspace::move(int step)//TODO erase step (just for tests)
{
  Vector s,c,a;  
  LeafContainer leafs = Octree::leafs;
  TemporaryContainer nb;
  //std::cout << " leaves "<< Octree::leafs.size() << std::endl;
  for (size_t i=0; i<leafs.size(); i++){
    Octree *it=leafs[i];
    nb.clear();
    (it)->returnNeighboursLeaf(nb);
    TemporaryContainer agentsleaf = (it)->agents;

    //Chargement mémoire sur GPU
    Agent *neighArray=tempToArray(nb);
    Agent *leafArray=tempToArray(leafs[i]->agents);
    Agent *d_neighArray;
    Agent *d_leafArray;
    //TODO penser à supprimer les liste d'agents copiés
    hipMalloc((void **)&d_neighArray,sizeof(Agent)*nb.size());
    hipMalloc((void **)&d_leafArray,sizeof(Agent)*leafs[i]->agents.size());
    hipMemcpy(d_neighArray,neighArray,sizeof(Agent)*nb.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_leafArray,leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyHostToDevice);

    //Initialiser la grille
    dim3 dimGrid(1,1,1);
    dim3 dimBlock(1,1,1);

    computeOnGPU<<<dimGrid,dimBlock>>>(0.25,d_leafArray);
    hipDeviceSynchronize();
    hipMemcpy(d_leafArray,leafArray,sizeof(Agent)*leafs[i]->agents.size(), hipMemcpyDeviceToHost);
    arrayToTemp(leafArray,leafs[i]->agents.size(),leafs[i]->agents);
      for(size_t j=0; j<agentsleaf.size(); j++){
       Agent *it2=agentsleaf[j];
      /* TemporaryContainer bufA,bufC,bufS;
       returnNeighboursBuffer(nb, it2,
        rCohesion, bufC,
        rAlignment, bufA,
        rSeparation, bufS);
       s = (it2)->separation(bufS, rSeparation);
       c = (it2)->cohesion(bufC, rCohesion);
       a = (it2)->alignment(bufA, rAlignment);
       (it2)->direction[1-Agent::curr_state] = wCohesion*c + wAlignment*a + wSeparation*s;

       (it2)->velocity[1-Agent::curr_state] = (it2)->velocity[Agent::curr_state] + (it2)->direction[1-Agent::curr_state];

       double speed = (it2)->velocity[1-Agent::curr_state].norm();
       if ((speed > maxU)) {
          (it2)->velocity[1-Agent::curr_state] = (it2)->velocity[1-Agent::curr_state] * maxU/speed;
      }

      (it2)->position[1-Agent::curr_state] = (it2)->position[Agent::curr_state] + dt*(it2)->velocity[Agent::curr_state];
*/
      (it2)->position[1-Agent::curr_state].x= fmod((it2)->position[1-Agent::curr_state].x,domainsize);
      (it2)->position[1-Agent::curr_state].y= fmod((it2)->position[1-Agent::curr_state].y,domainsize);
      (it2)->position[1-Agent::curr_state].z= fmod((it2)->position[1-Agent::curr_state].z,domainsize);

    }

    }

    //Agent::curr_state = 1 - Agent::curr_state;
    update();
}

void Workspace::returnNeighboursBuffer(TemporaryContainer &nb, Agent *agent,
  Real rc, TemporaryContainer &bufC,
  Real ra, TemporaryContainer &bufA,
  Real rs, TemporaryContainer &bufS
  ){
  for(int i=0; i<nb.size(); i++){
    Real dist =  (agent->position[Agent::curr_state] - nb[i]->position[Agent::curr_state]).norm();
    if(dist <= rc)
      bufC.push_back(agent);
    if(dist <= ra)
      bufA.push_back(agent);
    if(dist <= rs)
      bufS.push_back(agent);
  }  
}



void Workspace::update(){
  //#pragma omp parallel for
  LeafContainer leafs = Octree::leafs;

   for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++){
        if ((lf->position > lf->agents[j]->position[Agent::curr_state]) ||
         (lf->agents[j]->position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))){      
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          lf->agents[j]));
        //lf->agents.remove(&agents[k]);
        lf->delete_leaves();
        oc.add(*lf->agents[j]);
      } else {
         lf->agents[j]->leaf[Agent::curr_state]=lf;
      }   
    }
      }

  /*for(size_t k = 0; k< na; k++){
    if (Agent::curr_state)
    Octree *lf = agents[k].leaf[1-Agent::curr_state];
    //Retirer de la liste si nécessaire et rajouter au bon endroit
    if((lf->position > agents[k].position[Agent::curr_state]) 
      || (agents[k].position[Agent::curr_state] >= (lf->position + Vector(1,1,1)*lf->width))) {
        lf->agents.erase(std::find(lf->agents.begin(),
          lf->agents.end(),
          &agents[k]));
        //lf->agents.remove(&agents[k]);
        lf->delete_leaves();
        oc.add(agents[k]);
      } else {
        agents[k].leaf[Agent::curr_state]=lf;
      }
    }*/
  }


void Workspace::simulate(int nsteps) {
  // store initial position[Agent::curr_state]s
    save(0);

    // perform nsteps time steps of the simulation
    int step = 0;
    while (step++ < nsteps) {
    //std::cout << "coco" << step << std::endl; 
      this->move(step);
      //tst.printOctree(& this->oc);
      // store every 20 steps
      if (step%1 == 0) save(step);
    }
}

void Workspace::save(int stepid) {
  std::ofstream myfile;
  LeafContainer leafs=Octree::leafs;
  myfile.open("boids2.xyz", stepid==0 ? std::ios::out : std::ios::app);

    myfile << std::endl;
    myfile << na << std::endl;
    for (size_t i=0; i<leafs.size(); i++){
    Octree *lf=leafs[i];
      for (size_t j = 0; j < lf->agents.size(); j++)
        myfile << "B " << lf->agents[j]->position[Agent::curr_state];
}
    myfile.close();
}

